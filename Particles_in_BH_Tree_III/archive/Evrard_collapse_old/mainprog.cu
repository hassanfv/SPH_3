#include "hip/hip_runtime.h"
%%writefile test.cu

#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <random>
#include <chrono>
#include "bh_tree_iteration_v0.h"

using namespace std;





int main()
{

  int numParticles, numNodes;

  float *h_mass;
  float *h_x;
  float *h_y;
  float *h_z;
  float *h_ax;
  float *h_ay;
  float *h_az;

  int *h_child;
  int *h_start;
  int *h_sorted;
  int *h_count;

  float *d_left;
  float *d_right;
  float *d_bottom;
  float *d_top;
  float *d_front;
  float *d_back;

  float *dev_mass;
  float *dev_x;
  float *dev_y;
  float *dev_z;
  float *dev_ax;
  float *dev_ay;
  float *dev_az;

  int *d_index;
  int *d_child;
  int *d_start;
  int *d_sorted;
  int *d_count;

  int *d_mutex;  //used for locking 

  //int N = pow(2, 10);

  int blockSize_bh = blockSize;
  int gridSize_bh = (N + blockSize_bh - 1) / blockSize_bh;

  numParticles = nBodies; // nBodies is the number of patticles with Typ != -1.
  numNodes = 8 * numParticles + 15000;

  //int m = numNodes;

  // allocate host data
  h_mass = new float[numNodes];
  h_x = new float[numNodes];
  h_y = new float[numNodes];
  h_z = new float[numNodes];
  h_ax = new float[numNodes];
  h_ay = new float[numNodes];
  h_az = new float[numNodes];
  h_child = new int[8*numNodes];
  h_start = new int[numNodes];
  h_sorted = new int[numNodes];
  h_count = new int[numNodes];

  // allocate device data
  gpuErrchk(hipMalloc((void**)&d_left, sizeof(float)));
  gpuErrchk(hipMalloc((void**)&d_right, sizeof(float)));
  gpuErrchk(hipMalloc((void**)&d_bottom, sizeof(float)));
  gpuErrchk(hipMalloc((void**)&d_top, sizeof(float)));
  gpuErrchk(hipMalloc((void**)&d_front, sizeof(float)));
  gpuErrchk(hipMalloc((void**)&d_back, sizeof(float)));

  gpuErrchk(hipMemset(d_left, 0, sizeof(float)));
  gpuErrchk(hipMemset(d_right, 0, sizeof(float)));
  gpuErrchk(hipMemset(d_bottom, 0, sizeof(float)));
  gpuErrchk(hipMemset(d_top, 0, sizeof(float)));
  gpuErrchk(hipMemset(d_front, 0, sizeof(float)));
  gpuErrchk(hipMemset(d_back, 0, sizeof(float)));

  gpuErrchk(hipMalloc((void**)&dev_mass, numNodes*sizeof(float)));

  gpuErrchk(hipMalloc((void**)&dev_x, numNodes*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&dev_y, numNodes*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&dev_z, numNodes*sizeof(float)));

  gpuErrchk(hipMalloc((void**)&dev_ax, numNodes*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&dev_ay, numNodes*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&dev_az, numNodes*sizeof(float)));

  gpuErrchk(hipMalloc((void**)&d_index, sizeof(int)));
  gpuErrchk(hipMalloc((void**)&d_child, 8*numNodes*sizeof(int)));
  gpuErrchk(hipMalloc((void**)&d_start, numNodes*sizeof(int)));
  gpuErrchk(hipMalloc((void**)&d_sorted, numNodes*sizeof(int)));
  gpuErrchk(hipMalloc((void**)&d_count, numNodes*sizeof(int)));
  gpuErrchk(hipMalloc((void**)&d_mutex, sizeof(int))); 

  gpuErrchk(hipMemset(d_start, -1, numNodes*sizeof(int)));
  gpuErrchk(hipMemset(d_sorted, 0, numNodes*sizeof(int)));

  //int memSize = sizeof(float) * 2 * numParticles;
    
  reset_arrays_kernel<<< gridSize_bh, blockSize_bh >>>(d_mutex, dev_x, dev_y, dev_z, dev_mass, d_count, d_start, d_sorted, d_child, d_index,
                                                       d_left, d_right, d_bottom, d_top, d_front, d_back, numParticles, numNodes);
  hipDeviceSynchronize();

  // initializing x, y, z, mass -----
  hipMemcpy(x, d_x, N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(z, d_z, N * sizeof(float), hipMemcpyDeviceToHost);
    
  for (int i = 0; i < numParticles; i++)
  {
    h_x[i] = x[i];
    h_y[i] = y[i];
    h_z[i] = z[i];
    
    h_mass[i] = mass[i];
  }

  hipMemcpy(dev_x, h_x, numNodes * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_y, h_y, numNodes * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_z, h_z, numNodes * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_mass, h_mass, numNodes * sizeof(float), hipMemcpyHostToDevice);

  compute_bounding_box_kernel<<< gridSize_bh, blockSize_bh >>>(d_mutex, dev_x, dev_y, dev_z, d_left, d_right, d_bottom, d_top, d_front, d_back, numParticles);
  hipDeviceSynchronize();
  
  auto T_build_tree_kernel = std::chrono::high_resolution_clock::now();
  build_tree_kernel<<< 1, 256 >>>(dev_x, dev_y, dev_z, dev_mass, d_count, d_start, d_child, d_index, d_left, d_right, d_bottom, d_top, d_front, d_back,
                                  numParticles, numNodes);
  hipDeviceSynchronize();  
  auto end_build_tree_kernel = std::chrono::high_resolution_clock::now();
  auto elapsed_build_tree_kernel = std::chrono::duration_cast<std::chrono::nanoseconds>(end_build_tree_kernel - T_build_tree_kernel);
  cout << "Elapsed time = " << elapsed_build_tree_kernel.count() * 1e-9 << endl;
  
  
  centre_of_mass_kernel<<<gridSize_bh, blockSize_bh>>>(dev_x, dev_y, dev_z, dev_mass, d_index, numParticles);
  hipDeviceSynchronize();  
  
  
  sort_kernel<<< 1, 256 >>>(d_count, d_start, d_sorted, d_child, d_index, numParticles);
  hipDeviceSynchronize();  
  
  
  auto T_Force = std::chrono::high_resolution_clock::now();
  compute_forces_kernel<<< gridSize_bh, blockSize_bh >>>(dev_x, dev_y, dev_z, dev_ax, dev_ay, dev_az, dev_mass, d_sorted, d_child,
                                                         d_left, d_right, d_bottom, d_top, d_front, d_back, numParticles);
  hipDeviceSynchronize();
  auto end_Force = std::chrono::high_resolution_clock::now();
  auto elapsed_Force = std::chrono::duration_cast<std::chrono::nanoseconds>(end_Force - T_Force);
  cout << "T_Force = " << elapsed_Force.count() * 1e-9 << endl;
  
  
  hipMemcpy(h_ax, d_ax, numNodes * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_ay, d_ay, numNodes * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_az, d_az, numNodes * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < numParticles; i++)
  {
    //cout << "ax[" << i << "] = " << h_ax[i] << endl;
    cout << h_ay[i] << endl;
  }


}

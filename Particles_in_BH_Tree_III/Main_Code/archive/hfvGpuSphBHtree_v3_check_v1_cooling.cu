#include "hip/hip_runtime.h"
%%writefile test.cu
#include <iostream>
#include <fstream>
#include <cmath>
#include <string>
#include <vector>
#include <sstream>
#include <chrono>
#include <random>
#include <tuple>
#include "hfvCLibs_v7_hfv_v2.h"
#include "bh_tree_iteration_v2.h"
#include "ngb_v1.h"
#include <cstdlib> // This is ONLY used for the "exit(0)" function !!

// Added the isothermal gravitational field acceleration. (24 May 2023).
// Added the reading of the params.txt file and updated the IC reading file section and function. (22 May 2023).

using namespace std;

int main()
{

  float dt = 5e-7; //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! This is only the first time step !!

  const float Nngb_f = 64.0f; // used in smoothing func.
  const int Nngb = 64;
  const int Ndown = Nngb - 5;
  const int Nup = Nngb + 5;
  const float coeff = 0.005f; // used for smoothing length.
  
  const float kpc_in_cm = 3.086e21;
  
  //*******************************************************************
  //******************* Reading Cooling File **************************
  //*******************************************************************
  ifstream infile("coolHeatGridNew.bin", ios::binary);

  if (!infile) {
    cerr << "Failed to open coolHeatGridNew.bin file." << endl;
    return 1;
  }

  // Read the sizes
  int N_kpc, N_T, N_nH, N_Z, N_M, N_Time;
  infile.read(reinterpret_cast<char*>(&N_kpc), sizeof(int));
  infile.read(reinterpret_cast<char*>(&N_nH), sizeof(int));
  infile.read(reinterpret_cast<char*>(&N_Z), sizeof(int));
  infile.read(reinterpret_cast<char*>(&N_T), sizeof(int));
  infile.read(reinterpret_cast<char*>(&N_M), sizeof(int));
  infile.read(reinterpret_cast<char*>(&N_Time), sizeof(int));

  // Allocate and read the densities, temperatures, metallicities, and timeArr arrays
  vector<float> kpcArr(N_kpc);     // float
  vector<float> densities(N_nH);     // float
  vector<float> metallicities(N_Z);  // float
  vector<float> temperatures(N_T);   // float
  vector<float> timeArr(N_Time);     // float

  infile.read(reinterpret_cast<char*>(kpcArr.data()), N_kpc * sizeof(float));
  infile.read(reinterpret_cast<char*>(densities.data()), N_nH * sizeof(float));
  infile.read(reinterpret_cast<char*>(metallicities.data()), N_Z * sizeof(float));
  infile.read(reinterpret_cast<char*>(temperatures.data()), N_T * sizeof(float));
  infile.read(reinterpret_cast<char*>(timeArr.data()), N_Time * sizeof(float));

  // Allocate and read the flattened res and muArr array
  int N_HCool = N_kpc * N_T * N_nH * N_Z * N_Time;
  vector<float> res_flattened(N_HCool);  // float
  vector<float> muArr(N_HCool);  // float
  
  int N_metalz = N_kpc * N_T * N_nH * N_Z * N_M * N_Time;
  vector<float> metalzArr(N_metalz);  // float

  infile.read(reinterpret_cast<char*>(res_flattened.data()), N_HCool * sizeof(float));
  infile.read(reinterpret_cast<char*>(muArr.data()), N_HCool * sizeof(float)); // Note that muA and res_flattedned have the same structure!!
  
  infile.read(reinterpret_cast<char*>(metalzArr.data()), N_metalz * sizeof(float));

  infile.close();
  
  
//------------- Just for testing ---------
  int jjj = 2; // kpc
  int i = 20;   // T
  int j = 70;  // nH
  int k = 2;   // Z
  int l = 10;   // time
  
  int indx = jjj * (N_T * N_nH * N_Z * N_Time) + i * (N_nH * N_Z * N_Time) + j * (N_Z * N_Time) + k * N_Time + l;
  
  int ii_HI  = 0;
  int ii_HII = 1;
  int indx_HI  = jjj * (N_T * N_nH * N_Z * N_M * N_Time) + i * (N_nH * N_Z * N_M * N_Time) + j * (N_Z * N_M * N_Time) + k * (N_M * N_Time) + ii_HI * (N_Time) + l;
  int indx_HII = jjj * (N_T * N_nH * N_Z * N_M * N_Time) + i * (N_nH * N_Z * N_M * N_Time) + j * (N_Z * N_M * N_Time) + k * (N_M * N_Time) + ii_HII * (N_Time) + l;
  
  cout << "u = " << res_flattened[indx] << endl;
  cout << "mu = " << muArr[indx] << endl;
  cout << "HI fraction = " << metalzArr[indx_HI] << endl;
  cout << "HII fraction = " << metalzArr[indx_HII] << endl;
//--------------------------------------------
  
  float *Temp, *d_Temp, *nH, *d_nH, *Z, *d_Z, *Time, *d_Time, *HCool, *d_HCool, *muA, *d_muA, *kpc, *d_kpc, *metalz, *d_metalz;
  float *U, *d_U; // They will be used only inside the hcooling function as I could not define them inside the hcooling function (GPU climitations!!)
  
  kpc = new float[N_kpc];
  Temp = new float[N_T];
  nH = new float[N_nH];
  Z = new float[N_Z];
  Time = new float[N_Time];
  HCool = new float[N_HCool];
  muA = new float[N_HCool];
  U = new float[N_T];
  metalz = new float[N_metalz];
  
  hipMalloc(&d_kpc, N_kpc * sizeof(float));
  hipMalloc(&d_Temp, N_T * sizeof(float));
  hipMalloc(&d_nH, N_nH * sizeof(float));
  hipMalloc(&d_Z, N_Z * sizeof(float));
  hipMalloc(&d_Time, N_Time * sizeof(float));
  hipMalloc(&d_HCool, N_HCool * sizeof(float));
  hipMalloc(&d_muA, N_HCool * sizeof(float));
  hipMalloc(&d_U, N_T * sizeof(float));
  hipMalloc(&d_metalz, N_metalz * sizeof(float));
  
  for (int i = 0; i < N_kpc; i++)
  {
    kpc[i] = kpcArr[i];
  }
  
  for (int i = 0; i < N_T; i++)
  {
    Temp[i] = temperatures[i];
    U[i] = 0.0f;
  }
  
  for (int i = 0; i < N_nH; i++)
  {
    nH[i] = densities[i];
  }
  
  for (int i = 0; i < N_Z; i++)
  {
    Z[i] = metallicities[i];
  }
  
  for (int i = 0; i < N_Time; i++)
  {
    Time[i] = timeArr[i];
  }
  
  for (int i = 0; i < N_HCool; i++)
  {
    HCool[i] = res_flattened[i];
  }
  
  for (int i = 0; i < N_HCool; i++)
  {
    muA[i] = muArr[i];
  }

  for (int i = 0; i < N_metalz; i++)
  {
    metalz[i] = metalzArr[i];
  }

  // Copy from Host to Device
  hipMemcpy(d_kpc, kpc, N_kpc * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_Temp, Temp, N_T * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_nH, nH, N_nH * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_Z, Z, N_Z * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_Time, Time, N_Time * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_HCool, HCool, N_HCool * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_muA, muA, N_HCool * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_U, U, N_T * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_metalz, metalz, N_metalz * sizeof(float), hipMemcpyHostToDevice);

  //********************************************************************
  //**************** Reading the params.txt file ***********************
  //********************************************************************
  std::string filename;
  int N, ndx_BH;
  float GG, L_AGN_code_unit, M_dot_in, v_in, u_for_10K_Temp, m_sph_high_res, sigma, UnitDensity_in_cgs, Unit_u_in_cgs, unitTime_in_s,
        unitLength_in_cm;

  readParams(filename, N, ndx_BH, GG, L_AGN_code_unit, M_dot_in, v_in, u_for_10K_Temp, m_sph_high_res, sigma, UnitDensity_in_cgs, Unit_u_in_cgs, unitTime_in_s,
             unitLength_in_cm);

  std::cout << "filename: " << filename << "\n";
  std::cout << "N: " << N << "\n";
  std::cout << "ndx_BH: " << ndx_BH << "\n";
  std::cout << "GG: " << GG << "\n";
  std::cout << "L_AGN_code_unit: " << L_AGN_code_unit << "\n";
  std::cout << "M_dot_in_code_unit: " << M_dot_in << "\n";
  std::cout << "vin_in_code_unit: " << v_in << "\n";
  std::cout << "u_for_10K_Temp: " << u_for_10K_Temp << "\n";
  std::cout << "m_sph_high_res: " << m_sph_high_res << "\n";
  std::cout << "sigma: " << sigma << "\n";
  
  std::cout << "UnitDensity_in_cgs: " << UnitDensity_in_cgs << "\n";
  std::cout << "Unit_u_in_cgs: " << Unit_u_in_cgs << "\n";
  std::cout << "unitTime_in_s: " << unitTime_in_s << "\n";
  
  std::cout << "unitLength_in_cm: " << unitLength_in_cm << "\n";
  
  //*********************************************************************
  //******************** Reading the IC file ****************************
  //*********************************************************************
  auto data = readVectorsFromFile(filename);

  std::vector<int> &Typvec = std::get<0>(data);
  std::vector<float> &xvec = std::get<1>(data);
  std::vector<float> &yvec = std::get<2>(data);
  std::vector<float> &zvec = std::get<3>(data);
  std::vector<float> &vxvec = std::get<4>(data);
  std::vector<float> &vyvec = std::get<5>(data);
  std::vector<float> &vzvec = std::get<6>(data);
  std::vector<float> &uvec = std::get<7>(data);
  std::vector<float> &hvec = std::get<8>(data);
  std::vector<float> &epsvec = std::get<9>(data);
  std::vector<float> &massvec = std::get<10>(data);


  // declaring the arrays.
  int *Typ, *d_Typ;
  float *x, *d_x, *y, *d_y, *z, *d_z, *vx, *d_vx, *vy, *d_vy, *vz, *d_vz;
  float *mass, *d_mass, *h, *d_h, *rho, *d_rho;
  float *accx, *accy, *accz, *d_accx, *d_accy, *d_accz, *eps, *d_eps;
  float *P, *d_P, *csnd, *d_csnd, *divV, *d_divV, *curlV, *d_curlV;
  float *accx_sph, *accy_sph, *accz_sph, *d_accx_sph, *d_accy_sph, *d_accz_sph;
  float *accx_tot, *accy_tot, *accz_tot, *d_accx_tot, *d_accy_tot, *d_accz_tot;
  float *abs_acc_g, *abs_acc_tot, *v_sig, *dh_dt, *d_abs_acc_g, *d_abs_acc_tot;
  float *d_v_sig, *d_dh_dt, *u, *dudt, *d_u, *d_dudt, *utprevious;
  float *d_utprevious;
  float *Nngb_previous, *d_Nngb_previous; // Note that both are floats and not int! check smoothing func. to see why!
  float *dt_particles, *d_dt_particles;
  
  float *dudt_pre, *d_dudt_pre;

  float gammah = 5.0f / 3.0f;
  float GAMMA_MINUS1 = gammah - 1.0f;
  
  int N_ionFrac = N * N_M; // We have N_M species for each particle (N = total number of particles)
  
  float *ionFrac, *d_ionFrac;

  Typ = new int[N];

  x = new float[N];
  y = new float[N];
  z = new float[N];

  vx = new float[N];
  vy = new float[N];
  vz = new float[N];

  accx = new float[N];
  accy = new float[N];
  accz = new float[N];

  mass = new float[N];
  h = new float[N];
  rho = new float[N];
  eps = new float[N];
  P = new float[N];
  csnd = new float[N];

  divV = new float[N];
  curlV = new float[N];

  accx_sph = new float[N];
  accy_sph = new float[N];
  accz_sph = new float[N];

  accx_tot = new float[N];
  accy_tot = new float[N];
  accz_tot = new float[N];

  abs_acc_g = new float[N];
  abs_acc_tot = new float[N];
  v_sig = new float[N];
  dh_dt = new float[N];
  dt_particles = new float[N];

  u = new float[N];
  dudt = new float[N];
  utprevious = new float[N];
  
  dudt_pre = new float[N];

  Nngb_previous = new float[N];
  
  ionFrac = new float[N_ionFrac];

  hipMalloc(&d_Typ, N * sizeof(int));

  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));
  hipMalloc(&d_z, N * sizeof(float));

  hipMalloc(&d_vx, N * sizeof(float));
  hipMalloc(&d_vy, N * sizeof(float));
  hipMalloc(&d_vz, N * sizeof(float));

  hipMalloc(&d_accx, N * sizeof(float));
  hipMalloc(&d_accy, N * sizeof(float));
  hipMalloc(&d_accz, N * sizeof(float));

  hipMalloc(&d_mass, N * sizeof(float));
  hipMalloc(&d_h, N * sizeof(float));
  hipMalloc(&d_rho, N * sizeof(float));
  hipMalloc(&d_eps, N * sizeof(float));
  hipMalloc(&d_P, N * sizeof(float));
  hipMalloc(&d_csnd, N * sizeof(float));

  hipMalloc(&d_divV, N * sizeof(float));
  hipMalloc(&d_curlV, N * sizeof(float));

  hipMalloc(&d_accx_sph, N * sizeof(float));
  hipMalloc(&d_accy_sph, N * sizeof(float));
  hipMalloc(&d_accz_sph, N * sizeof(float));

  hipMalloc(&d_accx_tot, N * sizeof(float));
  hipMalloc(&d_accy_tot, N * sizeof(float));
  hipMalloc(&d_accz_tot, N * sizeof(float));

  hipMalloc(&d_abs_acc_g, N * sizeof(float));
  hipMalloc(&d_abs_acc_tot, N * sizeof(float));
  hipMalloc(&d_v_sig, N * sizeof(float));
  hipMalloc(&d_dh_dt, N * sizeof(float));
  hipMalloc(&d_dt_particles, N * sizeof(float));

  hipMalloc(&d_u, N * sizeof(float));
  hipMalloc(&d_dudt, N * sizeof(float));
  hipMalloc(&d_utprevious, N * sizeof(float));
  
  hipMalloc(&d_dudt_pre, N * sizeof(float));

  hipMalloc(&d_Nngb_previous, N * sizeof(float));
  
  hipMalloc(&d_ionFrac, N_ionFrac * sizeof(float));

  // Initialize x, y, z, etc on the Host.
  for (int i = 0; i < N; i++)
  {
    Typ[i] = Typvec[i];

    x[i] = xvec[i];
    y[i] = yvec[i];
    z[i] = zvec[i];

    vx[i] = vxvec[i];
    vy[i] = vyvec[i];
    vz[i] = vzvec[i];

    mass[i] = massvec[i];
    eps[i] = epsvec[i];

    accx[i] = 0.0f;
    accy[i] = 0.0f;
    accz[i] = 0.0f;

    accx_tot[i] = 0.0f;
    accy_tot[i] = 0.0f;
    accz_tot[i] = 0.0f;

    abs_acc_g[i] = 0.0f;
    abs_acc_tot[i] = 0.0f;
    v_sig[i] = 0.0f;

    h[i] = hvec[i]; // place holder.
    rho[i] = 0.0f;  // place holder.
    P[i] = 0.0f;    // placeholder.
    csnd[i] = 0.0f; // placeholder.

    divV[i] = 0.0f;  // placeholder.
    curlV[i] = 0.0f; // placeholder.

    accx_sph[i] = 0.0f;
    accy_sph[i] = 0.0f;
    accz_sph[i] = 0.0f;

    dh_dt[i] = 0.0f;

    u[i] = uvec[i];
    dudt[i] = 0.0f;
    utprevious[i] = 0.0f;
    
    dudt_pre[i] = 0.0f;

    dt_particles[i] = 0.0f;

    if (Typ[i] == 0)
    {
      Nngb_previous[i] = Nngb_f;
    }
    else
    {
      Nngb_previous[i] = 0.0f;
    }
  }
  
  for (int i = 0; i < N_ionFrac; i++)
  {
    ionFrac[i] = 0.0;
  }

  // Copy from Host to Device.
  hipMemcpy(d_Typ, Typ, N * sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_vx, vx, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_vy, vy, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_vz, vz, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_accx, accx, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_accy, accy, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_accz, accz, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_mass, mass, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_h, h, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_rho, rho, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_eps, eps, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_P, P, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_csnd, csnd, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_divV, divV, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_curlV, curlV, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_accx_sph, accx_sph, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_accy_sph, accy_sph, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_accz_sph, accz_sph, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_accx_tot, accx_tot, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_accy_tot, accy_tot, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_accz_tot, accz_tot, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_abs_acc_g, abs_acc_g, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_abs_acc_tot, abs_acc_tot, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_v_sig, v_sig, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dh_dt, dh_dt, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dt_particles, dt_particles, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_u, u, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dudt, dudt, N * sizeof(float), hipMemcpyHostToDevice);
  
  hipMemcpy(d_utprevious, utprevious, N * sizeof(float), hipMemcpyHostToDevice);
  
  hipMemcpy(d_dudt_pre, dudt_pre, N * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_Nngb_previous, Nngb_previous, N * sizeof(float), hipMemcpyHostToDevice);
  
  hipMemcpy(d_ionFrac, ionFrac, N_ionFrac * sizeof(float), hipMemcpyHostToDevice);
  
  
  // allocate device data
  float *d_leftX;
  float *d_rightX;
  float *d_bottomX;
  float *d_topX;
  float *d_frontX;
  float *d_backX;
  hipMalloc((void**)&d_leftX, sizeof(float));
  hipMalloc((void**)&d_rightX, sizeof(float));
  hipMalloc((void**)&d_bottomX, sizeof(float));
  hipMalloc((void**)&d_topX, sizeof(float));
  hipMalloc((void**)&d_frontX, sizeof(float));
  hipMalloc((void**)&d_backX, sizeof(float));

  hipMemset(d_leftX, 0, sizeof(float));
  hipMemset(d_rightX, 0, sizeof(float));
  hipMemset(d_bottomX, 0, sizeof(float));
  hipMemset(d_topX, 0, sizeof(float));
  hipMemset(d_frontX, 0, sizeof(float));
  hipMemset(d_backX, 0, sizeof(float));
  
  
  
  //============== For BH Tree =================
  int numParticles, numNodes;

  float *h_mass;
  float *h_x;
  float *h_y;
  float *h_z;
  float *h_ax;
  float *h_ay;
  float *h_az;

  int *h_child;
  int *h_start;
  int *h_sorted;
  int *h_count;

  float *d_left;
  float *d_right;
  float *d_bottom;
  float *d_top;
  float *d_front;
  float *d_back;

  float *dev_mass;
  float *dev_x;
  float *dev_y;
  float *dev_z;
  float *dev_ax;
  float *dev_ay;
  float *dev_az;

  int *d_index;
  int *d_child;
  int *d_start;
  int *d_sorted;
  int *d_count;

  int *d_mutex;  //used for locking
  
  int blockSize_bh, gridSize_bh;
  int nBodies;
  //===================
    
  

  //int blockSize = 256;                            // number of threads in a block
  int gridSize = (N + blockSize - 1) / blockSize; // Number of blocks in a grid

  const float visc_alpha = 1.0f;

  float t;

  t = 0.0f;

  float tEnd = 1.0f;
  float Nt = ceil(tEnd / dt) + 1;

  float Zmetal = 0.1; // ==> [Z/H] = -1.
  
  
  
  
  //------- For ngb_new -----
  int nSplit = 10; // for 3D, never go above 10 as 10^3 = 1000 and the maximum blockSize in GPUs is 1024!! IMPORTANT !!
  int Ncell = nSplit * nSplit * nSplit;
  
  //---- Finding nBodies (i.e. Those with Typ == 0 EXCLUDING BHole) REPEAT in each time-step -----
  int nBodiesB = 0;
  for (int i = 0; i < N; i++)
  {
    if (Typ[i] == 0) // BH should be excluded !!!
      nBodiesB++;
  }
  
  int *groupedIndex, *d_groupedIndex, *countx, *d_countx;
  
  groupedIndex = new int[nBodiesB];
  countx = new int[Ncell];
  
  hipMalloc((void **)&d_groupedIndex, nBodiesB * sizeof(int));
  hipMalloc((void **)&d_countx, Ncell * sizeof(int));

  for (int i = 0; i < nBodiesB; i++)
  {
    groupedIndex[i] = 0; // just a place holder. Its initial values don't matter as it will be replaced anyway!
  }
  
  for (int i = 0; i < (Ncell); i++)
  {
    countx[i] = 0;
  }
  
  hipMemcpy(d_groupedIndex, groupedIndex, nBodiesB * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_countx, countx, Ncell * sizeof(int), hipMemcpyHostToDevice);


  //----- computing maxCoord ------
  hipMemcpy(Typ, d_Typ, N * sizeof(int), hipMemcpyDeviceToHost);
  nBodies = 0;
  for (int i = 0; i < N; i++)
  {
    if (Typ[i] != -1)
    {
      nBodies++;
    }
  }
  int *d_mutexX;
  hipMalloc((void**)&d_mutexX, sizeof(int));
  gpuErrchk(hipMemset(d_mutexX, 0, sizeof(int)));
  compute_bounding_box_kernel<<< 1, blockSize >>>(d_mutexX, d_x, d_y, d_z, d_leftX, d_rightX, d_bottomX, d_topX, d_frontX, d_backX,
                                                  nBodies);
  hipDeviceSynchronize();
  
  float *h_leftX;
  float *h_rightX;
  float *h_bottomX;
  float *h_topX;
  float *h_frontX;
  float *h_backX;
  // allocate host data
  h_leftX = new float;
  h_rightX = new float;
  h_bottomX = new float;
  h_topX = new float;
  h_frontX = new float;
  h_backX = new float;
  
  hipMemcpy(h_leftX, d_leftX, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_rightX, d_rightX, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_bottomX, d_bottomX, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_topX, d_topX, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_frontX, d_frontX, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_backX, d_backX, sizeof(float), hipMemcpyDeviceToHost);
  
  printf("\n");
  printf("h_left, h_right, h_bottom, h_top, h_front, h_back = %f, %f, %f, %f, %f, %f\n",h_leftX[0],h_rightX[0], h_bottomX[0], h_topX[0], h_frontX[0], h_backX[0]);
  printf("\n");
  printf("\n");
  
  float maxRange = fmax(*h_leftX, fmax(*h_rightX, fmax(*h_bottomX, fmax(*h_topX, fmax(*h_frontX, *h_backX)))));
  float W_cell = ceil(2.0 * maxRange) / nSplit;
  
  float x_min = -1.0 * maxRange;
  float y_min = -1.0 * maxRange;
  float z_min = -1.0 * maxRange;
  
  ngbFinder<<<1, 1024, 2 * Ncell * sizeof(int)>>>(d_x, d_y, d_z, d_groupedIndex, d_countx, x_min, y_min, z_min, W_cell, nSplit, Ncell, nBodiesB);
  hipDeviceSynchronize();
  
  hipMemcpy(countx, d_countx, (Ncell) * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(groupedIndex, d_groupedIndex, nBodiesB * sizeof(int), hipMemcpyDeviceToHost);
  
  int *offSet;
  offSet = new int[Ncell+1];
  
  offSet[0] = 0;
  for (int i = 0; i < Ncell; i++)
  {
    offSet[i+1] = countx[i];
  }
  
  int *d_offSet;
  hipMalloc((void **)&d_offSet, (Ncell+1) * sizeof(int));
  hipMemcpy(d_offSet, offSet, (Ncell+1) * sizeof(int), hipMemcpyHostToDevice);
  //---------------------------
  
  //----------
  int MAX_ngb = 200;
  int MAX_N = N * MAX_ngb;
  int *ngb, *d_ngb;
  
  ngb = new int[MAX_N];
  
  hipMalloc(&d_ngb, MAX_N * sizeof(int));
  
  for (int i = 0; i < MAX_N; i++)
  {
    ngb[i] = -1;
  }

  hipMemcpy(d_ngb, ngb, MAX_N * sizeof(int), hipMemcpyHostToDevice);
  //----------
  
  //---------- ngbDebug
  int *ngbDebug, *d_ngbDebug;
  ngbDebug = new int[N]; // NOTE that for each particle we have one value!

  hipMalloc(&d_ngbDebug, N * sizeof(int));

  for (int i = 0; i < N; i++)
  {
    ngbDebug[i] = -1;
  }

  hipMemcpy(d_ngbDebug, ngbDebug, N * sizeof(int), hipMemcpyHostToDevice);
  //----------

  //------------------------------------------------------
  //------------------- ngbDB_new_v1 ---------------------
  //------------------------------------------------------
  auto T_ngb_new = std::chrono::high_resolution_clock::now();
  ngbDB_new_v1<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_h,
                                        x_min, y_min, z_min, W_cell, nSplit,
                                        d_offSet, d_groupedIndex,
                                        d_ngb, MAX_ngb, N);
  hipDeviceSynchronize();
  auto end_ngb_new = std::chrono::high_resolution_clock::now();
  auto elapsed_ngb_new = std::chrono::duration_cast<std::chrono::nanoseconds>(end_ngb_new - T_ngb_new);
  cout << "T_ngb_new = " << elapsed_ngb_new.count() * 1e-9 << endl;
  
  //-----------------------------------------------
  //-------------- Smoothing Length_ngb ---------------
  //-----------------------------------------------
  auto T_hh = std::chrono::high_resolution_clock::now();
  smoothing_h_ngb<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_h,
                                           N, Ndown, Nup, coeff,
                                           Nngb_f, d_Nngb_previous, d_divV, d_ngb, MAX_ngb, d_ngbDebug, dt);
  hipDeviceSynchronize();
  auto end_hh = std::chrono::high_resolution_clock::now();
  auto elapsed_hh = std::chrono::duration_cast<std::chrono::nanoseconds>(end_hh - T_hh);
  cout << "T_h = " << elapsed_hh.count() * 1e-9 << endl;
  
  //-----------------------------------------------
  //----------------- getDensity ------------------
  //-----------------------------------------------
  getDensity_ngb<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_mass,
                                          d_rho, d_h, d_ngb, MAX_ngb, N);
  hipDeviceSynchronize();

  //-----------------------------------------------
  //---------------- getPressure ------------------
  //-----------------------------------------------
  getPressure_Adiabatic<<<gridSize, blockSize>>>(d_Typ, d_P, d_rho, d_u, gammah, N);
  hipDeviceSynchronize();

  //-----------------------------------------------
  //----------------- getCsound -------------------
  //-----------------------------------------------
  getCsound_Adiabatic<<<gridSize, blockSize>>>(d_Typ, d_csnd, d_u, gammah, N);
  hipDeviceSynchronize();

  //-----------------------------------------------
  //----------------- div_curlV -------------------
  //-----------------------------------------------
  div_curlVel_ngb<<<gridSize, blockSize>>>(d_Typ, d_divV, d_curlV, d_x, d_y, d_z, d_vx, d_vy, d_vz,
                                           d_rho, d_mass, d_ngb, MAX_ngb, d_h, N);
  hipDeviceSynchronize();
  
  //-----------------------------------------------
  //------------------ acc_sph --------------------
  //-----------------------------------------------
  acc_sph_ngb<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_vx, d_vy, d_vz, d_h, d_csnd, d_rho,
                                       d_divV, d_curlV, d_mass, d_P, d_accx_sph, d_accy_sph,
                                       d_accz_sph, d_ngb, MAX_ngb, visc_alpha, N);
  hipDeviceSynchronize();


  //-----------------------------------------------
  //------------------ getAcc_g -------------------
  //-----------------------------------------------
  hipMemcpy(Typ, d_Typ, N * sizeof(int), hipMemcpyDeviceToHost);
  nBodies = 0;
  for (int i = 0; i < N; i++)
  {
    if (Typ[i] != -1)
    {
      nBodies++;
    }
  }
  
  cout << "nBodies = " << nBodies << endl;
  
  numParticles = nBodies; // nBodies is the number of patticles with Typ != -1.
  int eXtraSpace = 40000000;
  numNodes = 8 * numParticles + eXtraSpace;

  blockSize_bh = blockSize;
  gridSize_bh = (numParticles + blockSize_bh - 1) / blockSize_bh;
  
  // allocate host data
  h_mass = new float[numNodes];
  h_x = new float[numNodes];
  h_y = new float[numNodes];
  h_z = new float[numNodes];
  h_ax = new float[numNodes];
  h_ay = new float[numNodes];
  h_az = new float[numNodes];
  h_child = new int[8*numNodes];
  h_start = new int[numNodes];
  h_sorted = new int[numNodes];
  h_count = new int[numNodes];

  // allocate device data
  gpuErrchk(hipMalloc((void**)&d_left, sizeof(float)));
  gpuErrchk(hipMalloc((void**)&d_right, sizeof(float)));
  gpuErrchk(hipMalloc((void**)&d_bottom, sizeof(float)));
  gpuErrchk(hipMalloc((void**)&d_top, sizeof(float)));
  gpuErrchk(hipMalloc((void**)&d_front, sizeof(float)));
  gpuErrchk(hipMalloc((void**)&d_back, sizeof(float)));

  gpuErrchk(hipMemset(d_left, 0, sizeof(float)));
  gpuErrchk(hipMemset(d_right, 0, sizeof(float)));
  gpuErrchk(hipMemset(d_bottom, 0, sizeof(float)));
  gpuErrchk(hipMemset(d_top, 0, sizeof(float)));
  gpuErrchk(hipMemset(d_front, 0, sizeof(float)));
  gpuErrchk(hipMemset(d_back, 0, sizeof(float)));

  gpuErrchk(hipMalloc((void**)&dev_mass, numNodes*sizeof(float)));

  gpuErrchk(hipMalloc((void**)&dev_x, numNodes*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&dev_y, numNodes*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&dev_z, numNodes*sizeof(float)));

  gpuErrchk(hipMalloc((void**)&dev_ax, numNodes*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&dev_ay, numNodes*sizeof(float)));
  gpuErrchk(hipMalloc((void**)&dev_az, numNodes*sizeof(float)));

  gpuErrchk(hipMalloc((void**)&d_index, sizeof(int)));
  gpuErrchk(hipMalloc((void**)&d_child, 8*numNodes*sizeof(int)));
  gpuErrchk(hipMalloc((void**)&d_start, numNodes*sizeof(int)));
  gpuErrchk(hipMalloc((void**)&d_sorted, numNodes*sizeof(int)));
  gpuErrchk(hipMalloc((void**)&d_count, numNodes*sizeof(int)));
  gpuErrchk(hipMalloc((void**)&d_mutex, sizeof(int))); 

  gpuErrchk(hipMemset(d_start, -1, numNodes*sizeof(int)));
  gpuErrchk(hipMemset(d_sorted, 0, numNodes*sizeof(int)));
  gpuErrchk(hipMemset(d_mutex, 0, sizeof(int)));
 
  reset_arrays_kernel<<< 1, blockSize_bh >>>(d_mutex, dev_x, dev_y, dev_z, dev_mass, d_count, d_start, d_sorted, d_child, d_index,
                                             d_left, d_right, d_bottom, d_top, d_front, d_back, numParticles, numNodes);
  hipDeviceSynchronize();
  
  // initializing x, y, z, mass -----
  hipMemcpy(x, d_x, N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(z, d_z, N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(mass, d_mass, N * sizeof(float), hipMemcpyDeviceToHost);
    
  for (int i = 0; i < numParticles; i++)
  {
    h_x[i] = x[i];
    h_y[i] = y[i];
    h_z[i] = z[i];
    
    h_mass[i] = mass[i];
  }

  hipMemcpy(dev_x, h_x, numNodes * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_y, h_y, numNodes * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_z, h_z, numNodes * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_mass, h_mass, numNodes * sizeof(float), hipMemcpyHostToDevice);
  
  compute_bounding_box_kernel<<< 1, blockSize_bh >>>(d_mutex, dev_x, dev_y, dev_z, d_left, d_right, d_bottom, d_top, d_front, d_back,
                                                     numParticles);
  hipDeviceSynchronize();
  
  float *h_left;
  float *h_right;
  float *h_bottom;
  float *h_top;
  float *h_front;
  float *h_back;
  // allocate host data
  h_left = new float;
  h_right = new float;
  h_bottom = new float;
  h_top = new float;
  h_front = new float;
  h_back = new float;
  
  hipMemcpy(h_left, d_left, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_right, d_right, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_bottom, d_bottom, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_top, d_top, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_front, d_front, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_back, d_back, sizeof(float), hipMemcpyDeviceToHost);
  
  int *h_index = new int;
  hipMemcpy(h_index, d_index, sizeof(int), hipMemcpyDeviceToHost);
  printf("\n");
  printf("h_left, h_right, h_bottom, h_top, h_front, h_back = %f, %f, %f, %f, %f, %f\n", h_left[0], h_right[0], h_bottom[0], h_top[0], h_front[0], h_back[0]);
  printf("\n");
  printf("initial index = %d\n", h_index[0]);
  printf("\n");

  auto T_build_tree_kernel = std::chrono::high_resolution_clock::now();
  build_tree_kernel<<< 1, 256 >>>(dev_x, dev_y, dev_z, dev_mass, d_count, d_start, d_child, d_index, d_left, d_right, d_bottom, d_top, d_front, d_back,
                                  numParticles, numNodes);
  hipDeviceSynchronize();  
  auto end_build_tree_kernel = std::chrono::high_resolution_clock::now();
  auto elapsed_build_tree_kernel = std::chrono::duration_cast<std::chrono::nanoseconds>(end_build_tree_kernel - T_build_tree_kernel);
  cout << "Tree construction time = " << elapsed_build_tree_kernel.count() * 1e-9 << endl;
  
  
  //-------- Used for initial assessment to check if "numNodes" is big enough ---------
  hipMemcpy(h_child, d_child, 8 * numNodes * sizeof(int), hipMemcpyDeviceToHost);
  std::ofstream file("h_child.bin", std::ios::out | std::ios::binary);
  if (!file)
  {
      std::cerr << "Cannot open file for writing." << std::endl;
  }
  file.write(reinterpret_cast<char*>(h_child), 8 * numNodes * sizeof(int));
  file.close();
  cout << "Assessment file created !!!!" << endl;
  //-----------------------------------------------------------------------------------
  
  centre_of_mass_kernel<<<1, blockSize_bh>>>(dev_x, dev_y, dev_z, dev_mass, d_index, numParticles);
  hipDeviceSynchronize();  
  
  sort_kernel<<< 1, 256 >>>(d_count, d_start, d_sorted, d_child, d_index, numParticles);
  hipDeviceSynchronize();
  
  auto T_Force = std::chrono::high_resolution_clock::now();
  compute_forces_kernel<<< gridSize_bh, blockSize_bh >>>(dev_x, dev_y, dev_z, dev_ax, dev_ay, dev_az, dev_mass, d_eps, d_sorted, d_child,
                                                         d_left, d_right, d_bottom, d_top, d_front, d_back, numParticles);
  hipDeviceSynchronize();
  auto end_Force = std::chrono::high_resolution_clock::now();
  auto elapsed_Force = std::chrono::duration_cast<std::chrono::nanoseconds>(end_Force - T_Force);
  cout << "T_Force = " << elapsed_Force.count() * 1e-9 << endl;
  
  hipMemcpy(h_ax, dev_ax, numNodes * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_ay, dev_ay, numNodes * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(h_az, dev_az, numNodes * sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 0; i < numParticles; i++)
  {
    accx[i] = h_ax[i];
    accy[i] = h_ay[i];
    accz[i] = h_az[i];
  }
  
  hipMemcpy(d_accx, accx, numParticles * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_accy, accy, numParticles * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_accz, accz, numParticles * sizeof(float), hipMemcpyHostToDevice);
  
  delete[] h_mass;
  delete[] h_x;
  delete[] h_y;
  delete[] h_z;
  delete[] h_ax;
  delete[] h_ay;
  delete[] h_az;
  delete[] h_child;
  delete[] h_start;
  delete[] h_sorted;
  delete[] h_count;
  
  hipFree(d_left);
  hipFree(d_right);
  hipFree(d_bottom);
  hipFree(d_top);
  hipFree(d_front);
  hipFree(d_back);
  
  hipFree(dev_mass);
  hipFree(dev_x);
  hipFree(dev_y);
  hipFree(dev_z);
  
  hipFree(dev_ax);
  hipFree(dev_ay);
  hipFree(dev_az);
  
  hipFree(d_index);
  hipFree(d_child);
  hipFree(d_start);
  hipFree(d_sorted);
  hipFree(d_count);
  hipFree(d_mutex);

  //-----------------------------------------------
  //------------------ acc_tot --------------------
  //-----------------------------------------------
  acc_g_sph<<<gridSize, blockSize>>>(d_Typ, d_accx_tot, d_accy_tot, d_accz_tot,
                                     d_accx, d_accy, d_accz,
                                     d_accx_sph, d_accy_sph, d_accz_sph,
                                     N);
  hipDeviceSynchronize();

  //-----------------------------------------------
  //------------------- du_dt ---------------------
  //-----------------------------------------------  
  get_dU_ngb<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_vx, d_vy, d_vz, d_h, d_csnd, d_rho,
                                      d_divV, d_curlV, d_mass, d_P, d_dudt,
                                      d_ngb, MAX_ngb, visc_alpha, N);
  hipDeviceSynchronize();
  
  

  //-----------------------------------------------
  //---------------- u evolution ------------------
  //-----------------------------------------------

  u_updater<<<gridSize, blockSize>>>(d_Typ, d_u, d_dudt, d_utprevious, dt, N);
  hipDeviceSynchronize();

  //const float C_CFL = 0.25;
  
  delete[] ngb;
  hipFree(d_ngb);
  
  delete[] ngbDebug;
  hipFree(d_ngbDebug);
  
  delete[] groupedIndex;
  delete[] countx;
  delete[] offSet;
  
  hipFree(d_groupedIndex);
  hipFree(d_countx);
  hipFree(d_offSet);

  float leftover_mass = 0.0f;
  float *d_leftover_mass;
  hipMalloc((void **)&d_leftover_mass, sizeof(float));
  hipMemcpy(d_leftover_mass, &leftover_mass, sizeof(float), hipMemcpyHostToDevice);

  // **************************************************************
  // *********************** MAIN LOOP ****************************
  // **************************************************************

  int counter = 0; // This is used to save fewer output files, e.g. 1 snap-shot per 2 time-step!

  while (t < tEnd)
  {
  
    auto begin = std::chrono::high_resolution_clock::now();
  
    //----------
    int *ngb, *d_ngb;
    ngb = new int[MAX_N];

    hipMalloc(&d_ngb, MAX_N * sizeof(int));

    for (int i = 0; i < MAX_N; i++)
    {
      ngb[i] = -1;
    }

    auto T_MovingData_1 = std::chrono::high_resolution_clock::now();
    hipMemcpy(d_ngb, ngb, MAX_N * sizeof(int), hipMemcpyHostToDevice);
    auto end_MovingData_1 = std::chrono::high_resolution_clock::now();
    auto elapsed_MovingData_1 = std::chrono::duration_cast<std::chrono::nanoseconds>(end_MovingData_1 - T_MovingData_1);
    cout << "T_MovingData_1 = " << elapsed_MovingData_1.count() * 1e-9 << endl;
    //----------
    
    //---------- ngbDebug
    int *ngbDebug, *d_ngbDebug;
    ngbDebug = new int[N];

    hipMalloc(&d_ngbDebug, N * sizeof(int));

    for (int i = 0; i < N; i++)
    {
      ngbDebug[i] = -1;
    }

    auto T_MovingData_2 = std::chrono::high_resolution_clock::now();
    hipMemcpy(d_ngbDebug, ngbDebug, N * sizeof(int), hipMemcpyHostToDevice);
    auto end_MovingData_2 = std::chrono::high_resolution_clock::now();
    auto elapsed_MovingData_2 = std::chrono::duration_cast<std::chrono::nanoseconds>(end_MovingData_2 - T_MovingData_2);
    cout << "T_MovingData_2 = " << elapsed_MovingData_2.count() * 1e-9 << endl;
    //----------

    //****************** velocity evolution *******************
    v_evolve<<<gridSize, blockSize>>>(d_Typ, d_vx, d_vy, d_vz, d_accx_tot, d_accy_tot,
                                      d_accz_tot, dt, N);
    hipDeviceSynchronize();

    //****************** position evolution (BH fixed at [0, 0, 0]) *******************

    r_evolve<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_vx, d_vy, d_vz, dt, ndx_BH, N);
    hipDeviceSynchronize();
    
    
    //------- For ngb_new -----
    //---- Finding nBodies (i.e. Those with Typ == 0 EXCLUDING BHole) REPEAT in each time-step -----
    nBodiesB = 0;
    for (int i = 0; i < N; i++)
    {
      if (Typ[i] == 0) // BH should be excluded !!!
        nBodiesB++;
    }
    
    groupedIndex = new int[nBodiesB];
    countx = new int[Ncell];
    
    hipMalloc((void **)&d_groupedIndex, nBodiesB * sizeof(int));
    hipMalloc((void **)&d_countx, Ncell * sizeof(int));

    for (int i = 0; i < nBodiesB; i++)
    {
      groupedIndex[i] = 0; // just a place holder. Its initial values don't matter as it will be replaced anyway!
    }
    
    for (int i = 0; i < (Ncell); i++)
    {
      countx[i] = 0;
    }
    
    auto T_MovingData_3 = std::chrono::high_resolution_clock::now();
    hipMemcpy(d_groupedIndex, groupedIndex, nBodiesB * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_countx, countx, Ncell * sizeof(int), hipMemcpyHostToDevice);
    auto end_MovingData_3 = std::chrono::high_resolution_clock::now();
    auto elapsed_MovingData_3 = std::chrono::duration_cast<std::chrono::nanoseconds>(end_MovingData_3 - T_MovingData_3);
    cout << "T_MovingData_3 = " << elapsed_MovingData_3.count() * 1e-9 << endl;

    //----- computing maxCoord ------
    hipMemset(d_leftX, 0, sizeof(float));
    hipMemset(d_rightX, 0, sizeof(float));
    hipMemset(d_bottomX, 0, sizeof(float));
    hipMemset(d_topX, 0, sizeof(float));
    hipMemset(d_frontX, 0, sizeof(float));
    hipMemset(d_backX, 0, sizeof(float));
    
    hipMemcpy(Typ, d_Typ, N * sizeof(int), hipMemcpyDeviceToHost);
    nBodies = 0;
    for (int i = 0; i < N; i++)
    {
      if (Typ[i] != -1)
      {
        nBodies++;
      }
    }

    gpuErrchk(hipMemset(d_mutexX, 0, sizeof(int)));
    compute_bounding_box_kernel<<< 1, blockSize >>>(d_mutexX, d_x, d_y, d_z, d_leftX, d_rightX, d_bottomX, d_topX, d_frontX, d_backX,
                                                    nBodies);
    hipDeviceSynchronize();
    
    float maxRange = fmax(*h_leftX, fmax(*h_rightX, fmax(*h_bottomX, fmax(*h_topX, fmax(*h_frontX, *h_backX)))));
    W_cell = ceil(2.0 * maxRange) / nSplit;
    
    x_min = -1.0 * maxRange;
    y_min = -1.0 * maxRange;
    z_min = -1.0 * maxRange;
    
    ngbFinder<<<1, 1024, 2 * Ncell * sizeof(int)>>>(d_x, d_y, d_z, d_groupedIndex, d_countx, x_min, y_min, z_min, W_cell, nSplit, Ncell, nBodiesB);
    hipDeviceSynchronize();
    
    auto T_MovingData_4 = std::chrono::high_resolution_clock::now();
    hipMemcpy(countx, d_countx, (Ncell) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(groupedIndex, d_groupedIndex, nBodiesB * sizeof(int), hipMemcpyDeviceToHost);
    auto end_MovingData_4 = std::chrono::high_resolution_clock::now();
    auto elapsed_MovingData_4 = std::chrono::duration_cast<std::chrono::nanoseconds>(end_MovingData_4 - T_MovingData_4);
    cout << "T_MovingData_4 = " << elapsed_MovingData_4.count() * 1e-9 << endl;
    
    offSet = new int[Ncell+1];
    offSet[0] = 0;
    for (int i = 0; i < Ncell; i++)
    {
      offSet[i+1] = countx[i];
    }
    
    int *d_offSet;
    hipMalloc((void **)&d_offSet, (Ncell+1) * sizeof(int));
    hipMemcpy(d_offSet, offSet, (Ncell+1) * sizeof(int), hipMemcpyHostToDevice);
    //---------------------------

    //------------------------------------------------------
    //------------------- ngbDB_new_v1 ---------------------
    //------------------------------------------------------
    T_ngb_new = std::chrono::high_resolution_clock::now();
    ngbDB_new_v1<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_h,
                                          x_min, y_min, z_min, W_cell, nSplit,
                                          d_offSet, d_groupedIndex,
                                          d_ngb, MAX_ngb, N);
    hipDeviceSynchronize();
    end_ngb_new = std::chrono::high_resolution_clock::now();
    elapsed_ngb_new = std::chrono::duration_cast<std::chrono::nanoseconds>(end_ngb_new - T_ngb_new);
    cout << "T_ngb_new = " << elapsed_ngb_new.count() * 1e-9 << endl;
    
    //****************** Smoothing Length *********************
    auto T_hh = std::chrono::high_resolution_clock::now();
    smoothing_h_ngb<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_h,
                                             N, Ndown, Nup, coeff,
                                             Nngb_f, d_Nngb_previous, d_divV, d_ngb, MAX_ngb, d_ngbDebug, dt);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
    {
        printf("Error: %s\n", hipGetErrorString(err));
    }
    cout << "Smoothing Length ----> Successful!." << endl;
    auto end_hh = std::chrono::high_resolution_clock::now();
    auto elapsed_hh = std::chrono::duration_cast<std::chrono::nanoseconds>(end_hh - T_hh);
    cout << "T_h = " << elapsed_hh.count() * 1e-9 << endl;


    //****************** Set eps of Gas equal to h ******************
    set_eps_of_gas_to_h<<<gridSize, blockSize>>>(d_Typ, d_eps, d_h, N);
    hipDeviceSynchronize();

    //****************** getDensity ***********************
    auto T_density = std::chrono::high_resolution_clock::now();
    getDensity_ngb<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_mass,
                                            d_rho, d_h, d_ngb, MAX_ngb, N);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) 
    {
        printf("Error: %s\n", hipGetErrorString(err));
    }
    cout << "getDensity ----> Successful!." << endl;
    auto end_density = std::chrono::high_resolution_clock::now();
    auto elapsed_density = std::chrono::duration_cast<std::chrono::nanoseconds>(end_density - T_density);
    cout << "T_density = " << elapsed_density.count() * 1e-9 << endl;
    
    //****************** getPressure **********************
    getPressure_Adiabatic<<<gridSize, blockSize>>>(d_Typ, d_P, d_rho, d_u, gammah, N);
    hipDeviceSynchronize();

    //****************** getCsound ************************
    getCsound_Adiabatic<<<gridSize, blockSize>>>(d_Typ, d_csnd, d_u, gammah, N);
    hipDeviceSynchronize();

    //****************** div_curlVX ************************
    auto T_divCurl = std::chrono::high_resolution_clock::now();
    div_curlVel_ngb<<<gridSize, blockSize>>>(d_Typ, d_divV, d_curlV, d_x, d_y, d_z, d_vx, d_vy, d_vz,
                                             d_rho, d_mass, d_ngb, MAX_ngb, d_h, N);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) 
    {
        printf("Error: %s\n", hipGetErrorString(err));
    }
    cout << "divV ----> Successful!." << endl;
    auto end_divCurl = std::chrono::high_resolution_clock::now();
    auto elapsed_divCurl = std::chrono::duration_cast<std::chrono::nanoseconds>(end_divCurl - T_divCurl);
    cout << "T_divCurl = " << elapsed_divCurl.count() * 1e-9 << endl;
    
    //****************** acc_sphX **************************
    auto T_acc_sph = std::chrono::high_resolution_clock::now();
    acc_sph_ngb<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_vx, d_vy, d_vz, d_h, d_csnd, d_rho,
                                         d_divV, d_curlV, d_mass, d_P, d_accx_sph, d_accy_sph,
                                         d_accz_sph, d_ngb, MAX_ngb, visc_alpha, N);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) 
    {
        printf("Error: %s\n", hipGetErrorString(err));
    }
    cout << "acc_sph ----> Successful!." << endl;
    auto end_acc_sph = std::chrono::high_resolution_clock::now();
    auto elapsed_acc_sph = std::chrono::duration_cast<std::chrono::nanoseconds>(end_acc_sph - T_acc_sph);
    cout << "T_acc_sph = " << elapsed_acc_sph.count() * 1e-9 << endl;
    
    
    
    //****************** getAcc_gX *************************
    auto T_acc_g_Full_process = std::chrono::high_resolution_clock::now();
    hipMemcpy(Typ, d_Typ, N * sizeof(int), hipMemcpyDeviceToHost);
    nBodies = 0;
    for (int i = 0; i < N; i++)
    {
      if (Typ[i] != -1)
      {
        nBodies++;
      }
    }
    
    cout << "nBodies = " << nBodies << endl;
    
    numParticles = nBodies; // nBodies is the number of patticles with Typ != -1.
    numNodes = 8 * numParticles + eXtraSpace;

    blockSize_bh = blockSize;
    gridSize_bh = (numParticles + blockSize_bh - 1) / blockSize_bh;
    
    // allocate host data
    h_mass = new float[numNodes];
    h_x = new float[numNodes];
    h_y = new float[numNodes];
    h_z = new float[numNodes];
    h_ax = new float[numNodes];
    h_ay = new float[numNodes];
    h_az = new float[numNodes];
    h_child = new int[8*numNodes];
    h_start = new int[numNodes];
    h_sorted = new int[numNodes];
    h_count = new int[numNodes];

    // allocate device data
    gpuErrchk(hipMalloc((void**)&d_left, sizeof(float)));
    gpuErrchk(hipMalloc((void**)&d_right, sizeof(float)));
    gpuErrchk(hipMalloc((void**)&d_bottom, sizeof(float)));
    gpuErrchk(hipMalloc((void**)&d_top, sizeof(float)));
    gpuErrchk(hipMalloc((void**)&d_front, sizeof(float)));
    gpuErrchk(hipMalloc((void**)&d_back, sizeof(float)));

    gpuErrchk(hipMemset(d_left, 0, sizeof(float)));
    gpuErrchk(hipMemset(d_right, 0, sizeof(float)));
    gpuErrchk(hipMemset(d_bottom, 0, sizeof(float)));
    gpuErrchk(hipMemset(d_top, 0, sizeof(float)));
    gpuErrchk(hipMemset(d_front, 0, sizeof(float)));
    gpuErrchk(hipMemset(d_back, 0, sizeof(float)));

    gpuErrchk(hipMalloc((void**)&dev_mass, numNodes*sizeof(float)));

    gpuErrchk(hipMalloc((void**)&dev_x, numNodes*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&dev_y, numNodes*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&dev_z, numNodes*sizeof(float)));

    gpuErrchk(hipMalloc((void**)&dev_ax, numNodes*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&dev_ay, numNodes*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&dev_az, numNodes*sizeof(float)));

    gpuErrchk(hipMalloc((void**)&d_index, sizeof(int)));
    gpuErrchk(hipMalloc((void**)&d_child, 8*numNodes*sizeof(int)));
    gpuErrchk(hipMalloc((void**)&d_start, numNodes*sizeof(int)));
    gpuErrchk(hipMalloc((void**)&d_sorted, numNodes*sizeof(int)));
    gpuErrchk(hipMalloc((void**)&d_count, numNodes*sizeof(int)));
    gpuErrchk(hipMalloc((void**)&d_mutex, sizeof(int))); 

    gpuErrchk(hipMemset(d_start, -1, numNodes*sizeof(int)));
    gpuErrchk(hipMemset(d_sorted, 0, numNodes*sizeof(int)));
    gpuErrchk(hipMemset(d_mutex, 0, sizeof(int)));
   
    auto T_reset_kernel = std::chrono::high_resolution_clock::now();
    reset_arrays_kernel<<< 1, blockSize_bh >>>(d_mutex, dev_x, dev_y, dev_z, dev_mass, d_count, d_start, d_sorted, d_child, d_index,
                                               d_left, d_right, d_bottom, d_top, d_front, d_back, numParticles, numNodes);
    hipDeviceSynchronize();
    auto end_reset_kernel = std::chrono::high_resolution_clock::now();
    auto elapsed_reset_kernel = std::chrono::duration_cast<std::chrono::nanoseconds>(end_reset_kernel - T_reset_kernel);
    cout << "T_reset_kernel = " << elapsed_reset_kernel.count() * 1e-9 << endl;
    
    // initializing x, y, z, mass -----
    auto T_MovingData_5 = std::chrono::high_resolution_clock::now();
    hipMemcpy(x, d_x, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(z, d_z, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(mass, d_mass, N * sizeof(float), hipMemcpyDeviceToHost);
    auto end_MovingData_5 = std::chrono::high_resolution_clock::now();
    auto elapsed_MovingData_5 = std::chrono::duration_cast<std::chrono::nanoseconds>(end_MovingData_5 - T_MovingData_5);
    cout << "T_MovingData_5 = " << elapsed_MovingData_5.count() * 1e-9 << endl;
      
    for (int i = 0; i < numParticles; i++)
    {
      h_x[i] = x[i];
      h_y[i] = y[i];
      h_z[i] = z[i];
      
      h_mass[i] = mass[i];
    }

    auto T_MovingData_6 = std::chrono::high_resolution_clock::now();
    hipMemcpy(dev_x, h_x, numNodes * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_y, h_y, numNodes * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_z, h_z, numNodes * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_mass, h_mass, numNodes * sizeof(float), hipMemcpyHostToDevice);
    auto end_MovingData_6 = std::chrono::high_resolution_clock::now();
    auto elapsed_MovingData_6 = std::chrono::duration_cast<std::chrono::nanoseconds>(end_MovingData_6 - T_MovingData_6);
    cout << "T_MovingData_6 = " << elapsed_MovingData_6.count() * 1e-9 << endl;
    
    auto T_bounding_box = std::chrono::high_resolution_clock::now();
    compute_bounding_box_kernel<<< 1, blockSize_bh >>>(d_mutex, dev_x, dev_y, dev_z, d_left, d_right, d_bottom, d_top, d_front, d_back,
                                                                 numParticles);
    hipDeviceSynchronize();
    auto end_bounding_box = std::chrono::high_resolution_clock::now();
    auto elapsed_bounding_box = std::chrono::duration_cast<std::chrono::nanoseconds>(end_bounding_box - T_bounding_box);
    cout << "T_bounding_box = " << elapsed_bounding_box.count() * 1e-9 << endl;

    T_build_tree_kernel = std::chrono::high_resolution_clock::now();
    build_tree_kernel<<< 1, 256 >>>(dev_x, dev_y, dev_z, dev_mass, d_count, d_start, d_child, d_index, d_left, d_right, d_bottom, d_top, d_front, d_back,
                                    numParticles, numNodes);
    hipDeviceSynchronize();  
    end_build_tree_kernel = std::chrono::high_resolution_clock::now();
    elapsed_build_tree_kernel = std::chrono::duration_cast<std::chrono::nanoseconds>(end_build_tree_kernel - T_build_tree_kernel);
    cout << "Tree construction time = " << elapsed_build_tree_kernel.count() * 1e-9 << endl;
    
    centre_of_mass_kernel<<<1, blockSize_bh>>>(dev_x, dev_y, dev_z, dev_mass, d_index, numParticles);
    hipDeviceSynchronize();  
    
    auto T_sort = std::chrono::high_resolution_clock::now();
    sort_kernel<<< 1, 256 >>>(d_count, d_start, d_sorted, d_child, d_index, numParticles);
    hipDeviceSynchronize();
    auto end_sort = std::chrono::high_resolution_clock::now();
    auto elapsed_sort = std::chrono::duration_cast<std::chrono::nanoseconds>(end_sort - T_sort);
    cout << "T_sort = " << elapsed_sort.count() * 1e-9 << endl; 
    
    T_Force = std::chrono::high_resolution_clock::now();
    compute_forces_kernel<<< gridSize_bh, blockSize_bh >>>(dev_x, dev_y, dev_z, dev_ax, dev_ay, dev_az, dev_mass, d_eps, d_sorted, d_child,
                                                           d_left, d_right, d_bottom, d_top, d_front, d_back, numParticles);
    hipDeviceSynchronize();
    end_Force = std::chrono::high_resolution_clock::now();
    elapsed_Force = std::chrono::duration_cast<std::chrono::nanoseconds>(end_Force - T_Force);
    cout << "T_Force = " << elapsed_Force.count() * 1e-9 << endl;
    
    
    auto T_MovingData_7 = std::chrono::high_resolution_clock::now();
    hipMemcpy(h_ax, dev_ax, numNodes * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_ay, dev_ay, numNodes * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_az, dev_az, numNodes * sizeof(float), hipMemcpyDeviceToHost);
    auto end_MovingData_7 = std::chrono::high_resolution_clock::now();
    auto elapsed_MovingData_7 = std::chrono::duration_cast<std::chrono::nanoseconds>(end_MovingData_7 - T_MovingData_7);
    cout << "T_MovingData_7 = " << elapsed_MovingData_7.count() * 1e-9 << endl;

    for (int i = 0; i < numParticles; i++)
    {
      accx[i] = h_ax[i];
      accy[i] = h_ay[i];
      accz[i] = h_az[i];
    }
    
    auto T_MovingData_8 = std::chrono::high_resolution_clock::now();
    hipMemcpy(d_accx, accx, numParticles * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_accy, accy, numParticles * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_accz, accz, numParticles * sizeof(float), hipMemcpyHostToDevice);
    auto end_MovingData_8 = std::chrono::high_resolution_clock::now();
    auto elapsed_MovingData_8 = std::chrono::duration_cast<std::chrono::nanoseconds>(end_MovingData_8 - T_MovingData_8);
    cout << "T_MovingData_8 = " << elapsed_MovingData_8.count() * 1e-9 << endl;
    
    delete[] h_mass;
    delete[] h_x;
    delete[] h_y;
    delete[] h_z;
    delete[] h_ax;
    delete[] h_ay;
    delete[] h_az;
    delete[] h_child;
    delete[] h_start;
    delete[] h_sorted;
    delete[] h_count;
    
    hipFree(d_left);
    hipFree(d_right);
    hipFree(d_bottom);
    hipFree(d_top);
    hipFree(d_front);
    hipFree(d_back);
    
    hipFree(dev_mass);
    hipFree(dev_x);
    hipFree(dev_y);
    hipFree(dev_z);
    
    hipFree(dev_ax);
    hipFree(dev_ay);
    hipFree(dev_az);
    
    hipFree(d_index);
    hipFree(d_child);
    hipFree(d_start);
    hipFree(d_sorted);
    hipFree(d_count);
    hipFree(d_mutex);

    auto end_acc_g_Full_process = std::chrono::high_resolution_clock::now();
    auto elapsed_acc_g_Full_process = std::chrono::duration_cast<std::chrono::nanoseconds>(end_acc_g_Full_process - T_acc_g_Full_process);
    cout << "T_acc_g_Full_process = " << elapsed_acc_g_Full_process.count() * 1e-9 << endl;

    //****************** acc_tot **************************
    auto T_acc_tot = std::chrono::high_resolution_clock::now();
    acc_g_sph<<<gridSize, blockSize>>>(d_Typ, d_accx_tot, d_accy_tot, d_accz_tot,
                                       d_accx, d_accy, d_accz,
                                       d_accx_sph, d_accy_sph, d_accz_sph,
                                       N);
    hipDeviceSynchronize();
    auto end_acc_tot = std::chrono::high_resolution_clock::now();
    auto elapsed_acc_tot = std::chrono::duration_cast<std::chrono::nanoseconds>(end_acc_tot - T_acc_tot);
    cout << "T_acc_tot = " << elapsed_acc_tot.count() * 1e-9 << endl;

    
    //******* Isothermal Gravity (Richings et al - 2018) ********
    galaxy_isothermal_potential<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_accx_tot,
                                                         d_accy_tot, d_accz_tot, sigma, G, N);
    hipDeviceSynchronize();
    

    //****************** velocity evolution *******************
    v_evolve<<<gridSize, blockSize>>>(d_Typ, d_vx, d_vy, d_vz, d_accx_tot, d_accy_tot,
                                      d_accz_tot, dt, N);
    hipDeviceSynchronize();

    //******************** get_dUX (du_dt) *********************
    auto T_dU = std::chrono::high_resolution_clock::now();
    get_dU_ngb<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z, d_vx, d_vy, d_vz, d_h, d_csnd, d_rho,
                                      d_divV, d_curlV, d_mass, d_P, d_dudt,
                                      d_ngb, MAX_ngb, visc_alpha, N);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) 
    {
        printf("Error: %s\n", hipGetErrorString(err));
    }
    cout << "get_dU ----> Successful!." << endl;
    auto end_dU = std::chrono::high_resolution_clock::now();
    auto elapsed_dU = std::chrono::duration_cast<std::chrono::nanoseconds>(end_dU - T_dU);
    cout << "T_dU = " << elapsed_dU.count() * 1e-9 << endl;

    //******************** u evolution *********************
    
    u_updater<<<gridSize, blockSize>>>(d_Typ, d_u, d_dudt, d_utprevious, dt, N);
    hipDeviceSynchronize();
    
    
    
    //****************** Heating & Cooling ********************
    auto T_cool = std::chrono::high_resolution_clock::now();
    hcoolingx<<<gridSize, blockSize>>>(d_Typ, d_u, d_U, d_rho, d_metalz, Zmetal, dt, // Zmetal is the gass metallicity assumed.
                                      d_nH, d_Z, d_HCool, d_ionFrac, d_Time, d_x, d_y, d_z,
                                      d_muA, d_Temp, d_kpc, UnitDensity_in_cgs, unitTime_in_s, 
                                      Unit_u_in_cgs, unitLength_in_cm, kpc_in_cm, GAMMA_MINUS1,
                                      N_kpc, N_nH, N_Z, N_T, N_M, N_Time, N);
    hipDeviceSynchronize();
    auto end_cool = std::chrono::high_resolution_clock::now();
    auto elapsed_cool = std::chrono::duration_cast<std::chrono::nanoseconds>(end_cool - T_cool);
    cout << "T_cool = " << elapsed_cool.count() * 1e-9 << endl;
    
    
    //-------------------------------------------------

    hipMemcpy(rho, d_rho, N * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < 5; i++)
    {
      cout << "AAA = " << rho[i] << endl;
    }

    auto T_SaveFile = std::chrono::high_resolution_clock::now();
    //------------ SAVING SNAP-SHOTS ------------
    if (!(counter % 100))
    //if (counter > -1)
    {
      hipMemcpy(Typ, d_Typ, N * sizeof(int), hipMemcpyDeviceToHost);

      hipMemcpy(x, d_x, N * sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(z, d_z, N * sizeof(float), hipMemcpyDeviceToHost);

      hipMemcpy(vx, d_vx, N * sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(vy, d_vy, N * sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(vz, d_vz, N * sizeof(float), hipMemcpyDeviceToHost);

      hipMemcpy(rho, d_rho, N * sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(h, d_h, N * sizeof(float), hipMemcpyDeviceToHost);

      hipMemcpy(u, d_u, N * sizeof(float), hipMemcpyDeviceToHost);
      
      hipMemcpy(mass, d_mass, N * sizeof(float), hipMemcpyDeviceToHost);

      hipMemcpy(ionFrac, d_ionFrac, N_ionFrac * sizeof(float), hipMemcpyDeviceToHost);
      
      hipMemcpy(ngbDebug, d_ngbDebug, N * sizeof(int), hipMemcpyDeviceToHost);

      // Specify the output file name
      std::string filename = "./Outputs/G-" + to_string(t * 10) + ".bin";
      // Save the arrays to binary format
      saveArraysToBinary(filename, x, y, z, vx, vy, vz, rho, h, u, mass, ionFrac, Typ, N, N_ionFrac, ngbDebug);
    }
    auto end_SaveFile = std::chrono::high_resolution_clock::now();
    auto elapsed_SaveFile = std::chrono::duration_cast<std::chrono::nanoseconds>(end_SaveFile - T_SaveFile);
    cout << "T_SaveFile = " << elapsed_SaveFile.count() * 1e-9 << endl;


    //******************************************************
    //************* Updating Time-step dt ******************
    //******************************************************
    /*
    dt_array_indiv_dt<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z,
                                               d_vx, d_vy, d_vz,
                                               d_accx, d_accy, d_accz,
                                               d_accx_tot, d_accy_tot, d_accz_tot,
                                               d_h, d_csnd, d_dt_particles,
                                               d_abs_acc_g, d_abs_acc_tot,
                                               d_divV, d_dh_dt, C_CFL,
                                               visc_alpha, d_eps, N);
    hipDeviceSynchronize();

    hipMemcpy(dt_particles, d_dt_particles, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Typ, d_Typ, N * sizeof(int), hipMemcpyDeviceToHost);
    */

    t += dt;

    // dt = min_finder(Typ, dt_particles, N);

    //***********************************************************
    //*************** Outflow particle injection ****************
    //***********************************************************
    // Generate a seed using the high resolution clock
    auto now = std::chrono::high_resolution_clock::now();
    auto nanos = std::chrono::duration_cast<std::chrono::nanoseconds>(now.time_since_epoch()).count();
    unsigned long long seed = counter; //static_cast<unsigned long long>(nanos); //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //------------
    outflow_injector2<<<gridSize, blockSize>>>(d_Typ, d_x, d_y, d_z,
                                              d_vx, d_vy, d_vz,
                                              d_h, d_eps, d_mass,
                                              Nngb_f, d_Nngb_previous,
                                              d_u, M_dot_in, v_in,
                                              m_sph_high_res, u_for_10K_Temp,
                                              d_leftover_mass, dt, ndx_BH, N,
                                              seed);
    hipDeviceSynchronize();
    

    if (!(counter % 1))
    {
      cout << "Adopted dt = " << dt << endl;
      cout << "current t = " << t << endl;
      cout << "*****************************" << endl;
      cout << endl;
    }

    delete[] ngb;
    hipFree(d_ngb);
    delete[] ngbDebug;
    hipFree(d_ngbDebug);
    delete[] groupedIndex;
    delete[] countx;
    delete[] offSet;
    
    hipFree(d_groupedIndex);
    hipFree(d_countx);
    hipFree(d_offSet);

    counter++;
    
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);
    cout << "Elapsed time = " << elapsed.count() * 1e-9 << endl;
    cout << endl;
    
  }

  delete[] Typ;
  delete[] x;
  delete[] y;
  delete[] z;
  delete[] vx;
  delete[] vy;
  delete[] vz;
  delete[] mass;
  delete[] h;
  delete[] rho;
  delete[] accx;
  delete[] accy;
  delete[] accz;
  delete[] eps;
  delete[] P;
  delete[] csnd;
  delete[] divV;
  delete[] curlV;
  delete[] accx_sph;
  delete[] accy_sph;
  delete[] accz_sph;
  delete[] accx_tot;
  delete[] accy_tot;
  delete[] accz_tot;
  delete[] abs_acc_g;
  delete[] abs_acc_tot;
  delete[] v_sig;
  delete[] dh_dt;
  delete[] u;
  delete[] dudt;
  delete[] utprevious;

  hipFree(d_Typ);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_vx);
  hipFree(d_vy);
  hipFree(d_vz);
  hipFree(d_mass);
  hipFree(d_h);
  hipFree(d_rho);
  hipFree(d_accx);
  hipFree(d_accy);
  hipFree(d_accz);
  hipFree(d_P);
  hipFree(d_csnd);
  hipFree(d_divV);
  hipFree(d_curlV);
  hipFree(d_accx_sph);
  hipFree(d_accy_sph);
  hipFree(d_accz_sph);
  hipFree(d_accx_tot);
  hipFree(d_accy_tot);
  hipFree(d_accz_tot);
  hipFree(d_abs_acc_g);
  hipFree(d_abs_acc_tot);
  hipFree(d_v_sig);
  hipFree(d_dh_dt);
  hipFree(d_u);
  hipFree(d_dudt);
  hipFree(d_utprevious);
}
